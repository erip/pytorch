#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/Generator.h>
#include <ATen/Tensor.h>
#include <ATen/native/DistributionTemplates.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/core/MT19937RNGEngine.h>
#include <memory>

using namespace at;

struct CUDA_CSPRNG_GeneratorImpl : public at::CPUGeneratorImpl {
  CUDA_CSPRNG_GeneratorImpl(uint64_t seed_in = default_rng_seed_val) : CPUGeneratorImpl(seed_in) {
    this->key_set_ = DispatchKeySet(DispatchKey::CustomRNGKeyId);
  }
};

typedef ulonglong2 block_t;
constexpr size_t block_t_size = sizeof(block_t);

// =========================================================== AES ===========================================================

__device__ static uint8_t sbox[256] = {
    0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
    0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
    0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
    0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
    0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
    0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
    0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
    0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
    0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
    0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
    0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
    0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
    0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
    0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
    0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
    0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
 };

//reduce the size of this since we don't need this many
__device__ static uint8_t rcon[255] = {
  0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 
  0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 
  0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 
  0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 
  0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 
  0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 
  0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 
  0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 
  0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 
  0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 
  0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 
  0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 
  0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 
  0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 
  0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 
  0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb  };

__device__ void add_round_key(uint8_t *block, uint8_t *key){
  for (int i = 0; i < 16; ++i) {
    block[i] = block[i] ^ key[i];
  }
}

__device__ void mix_columns(uint8_t *block){
  for (int i = 0; i < 4; ++i) { //iterate over columns
    uint8_t a[4];
    uint8_t b[4]; 
    uint8_t h;
    for (int j = 0; j < 4; ++j) {
      a[j] = block[4*i + j];
      h = (uint8_t)((int8_t)a[j] >> 7);
      b[j] = a[j] << 1;
      b[j] ^= 0x1b & h;
    } 
    block[4*i + 0] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
    block[4*i + 1] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
    block[4*i + 2] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
    block[4*i + 3] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0]; 
  }
}

__device__ void sub_bytes(uint8_t *block){
  for (int i = 0; i < 16; ++i) {
    block[i] = sbox[block[i]];
  }
}

//shift left by 0,1,2,3 respectively 
__device__ void shift_rows(uint8_t *block) {
  uint8_t tmp;
  //row 0 remains unshifted

  //shift row 1 left by 1
  tmp = block[1];
  block[1] = block[5];
  block[5] = block[9];
  block[9] = block[13];
  block[13] = tmp;

  //shift row 2 letf by 2
  tmp = block[2];
  block[2] = block[10];
  block[10] = tmp;

  tmp = block[6];
  block[6] = block[14];
  block[14] = tmp;

  //shift row 3 left by 3
  tmp = block[3];
  block[3] = block[15];
  block[15] = block[11];
  block[11] = block[7];
  block[7] = tmp;
}

//generate round keys from initial key
__device__ void expand_key(uint8_t *key, uint8_t *rkey){
  uint32_t i,j,k;
  uint8_t tempa[4];
  uint32_t nround = 10;
  //first round key is just the key
  for (i = 0; i < 4; ++i) {
    rkey[4*i + 0] = key[4*i + 0];
    rkey[4*i + 1] = key[4*i + 1];
    rkey[4*i + 2] = key[4*i + 2];
    rkey[4*i + 3] = key[4*i + 3];
  }
  for (i = 4; i < 4*(nround + 1); ++i) {
    for (j = 0; j < 4; ++j) {
      tempa[j] = rkey[(i-1)*4 + j];
    }
    if (i % 4 == 0) {
      //rotate 4 bytes in word
      k = tempa[0];
      tempa[0] = tempa[1];
      tempa[1] = tempa[2];
      tempa[2] = tempa[3];
      tempa[3] = k;

      tempa[0] = sbox[tempa[0]];
      tempa[1] = sbox[tempa[1]];
      tempa[2] = sbox[tempa[2]];
      tempa[3] = sbox[tempa[3]];
  
      tempa[0] = tempa[0] ^ rcon[i/4];
    }
    rkey[4*i + 0] = rkey[4*(i-4) + 0] ^ tempa[0];
    rkey[4*i + 1] = rkey[4*(i-4) + 1] ^ tempa[1];
    rkey[4*i + 2] = rkey[4*(i-4) + 2] ^ tempa[2];
    rkey[4*i + 3] = rkey[4*(i-4) + 3] ^ tempa[3];
  } 
}

__device__ void encrypt(uint8_t *block, uint8_t *key) {
  uint8_t rkey[176];
  uint8_t round; 
  //setup round keys
  expand_key(key, rkey); 
  //perform encryption
  add_round_key(block, rkey);
  for(round = 1; round < 10; ++round){
    sub_bytes(block);
    shift_rows(block);
    mix_columns(block);
    add_round_key(block, rkey + 16*round);
  }
  sub_bytes(block);
  shift_rows(block);
  add_round_key(block, rkey + 16*round);
}

// ===========================================================================================================================

Tensor key_tensor(Generator generator) {
  return torch::empty({16}, torch::kUInt8).random_(0, 256, generator).to(kCUDA);
}

template<size_t size>
struct DummyRNG {
  __device__ DummyRNG(uint64_t* vals) {
    for (auto i = 0; i < size; i++) {
      vals_[i] = vals[i];
    }
  }
  uint32_t __device__ random() { return static_cast<uint32_t>(vals_[index++]); }
  uint64_t __device__ random64() { return vals_[index++]; }
  c10::optional<float> __device__ next_float_normal_sample() { return c10::nullopt; }
  c10::optional<double> __device__ next_double_normal_sample() { return c10::nullopt; }
  void __device__ set_next_float_normal_sample(c10::optional<float> randn) {}
  void __device__ set_next_double_normal_sample(c10::optional<double> randn) {}
private:
  uint64_t vals_[size];
  int index = 0;
};

template<typename scalar_t, typename uint_t, size_t N = 1, typename cipher_t, typename transform_t>
__global__ void block_cipher_contiguous_kernel(scalar_t* data, int numel, cipher_t cipher, transform_t transform_func) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  constexpr auto unroll_factor = block_t_size / sizeof(uint_t) / N;
  if (unroll_factor * idx < numel) {
    auto block = cipher(idx);
    #pragma unroll
    for (auto i = 0; i < unroll_factor; ++i) {
      const auto li = unroll_factor * idx + i;
      if (li < numel) {
        uint64_t vals[N];
        #pragma unroll
        for (auto j = 0; j < N; j++) {
          vals[j] = (reinterpret_cast<uint_t*>(&block))[N * i + j];
        }
        DummyRNG<N> rng(vals);
        data[li] = transform_func(&rng);
      }
    }
  }
}

template<typename scalar_t, typename uint_t, size_t N = 1, typename cipher_t, typename transform_t>
__global__ void block_cipher_kernel(scalar_t* data, int numel, cipher_t cipher, transform_t transform_func, OffsetCalculator<1> offset_calc) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  constexpr auto unroll_factor = block_t_size / sizeof(uint_t) / N;
  if (unroll_factor * idx < numel) {
    auto block = cipher(idx);
    #pragma unroll
    for (auto i = 0; i < unroll_factor; ++i) {
      const auto li = unroll_factor * idx + i;
      if (li < numel) {
        uint64_t vals[N];
        #pragma unroll
        for (auto j = 0; j < N; j++) {
          vals[j] = (reinterpret_cast<uint_t*>(&block))[N * i + j];
        }
        DummyRNG<N> rng(vals);
        data[offset_calc.get(li)[0] / sizeof(scalar_t)] = transform_func(&rng);
      }
    }
  }
}

template<typename scalar_t, typename uint_t, size_t N = 1, typename cipher_t, typename transform_t>
void block_cipher_ctr_mode(at::TensorIterator& iter, cipher_t cipher, transform_t transform_func) {
  const auto numel = iter.numel();
  if (numel == 0) {
    return;
  }
  constexpr auto unroll_factor = block_t_size / sizeof(uint_t) / N;
  const auto block = 256;
  const auto grid = (numel + (block * unroll_factor) - 1) / (block * unroll_factor);
  scalar_t* data = (scalar_t*)iter.data_ptr(0);
  auto stream = at::cuda::getCurrentCUDAStream();
  if (iter.output(0).is_contiguous()) {
    block_cipher_contiguous_kernel<scalar_t, uint_t, N, cipher_t, transform_t><<<grid, block, 0, stream>>>(data, numel, cipher, transform_func);
  } else {
    auto offset_calc = make_offset_calculator<1>(iter);
    block_cipher_kernel<scalar_t, uint_t, N, cipher_t, transform_t><<<grid, block, 0, stream>>>(data, numel, cipher, transform_func, offset_calc);
  }
  AT_CUDA_CHECK(hipGetLastError());
}

// ===========================================================================================================================

template<typename scalar_t, typename uint_t, size_t N = 1, typename transform_t>
void block_cipher_helper(TensorIterator& iter, uint8_t* key, transform_t transform_func) {
  block_cipher_ctr_mode<scalar_t, uint_t, N>(iter,
    [key] __device__ (unsigned int idx) -> block_t {
      block_t block;
      memset(&block, 0, block_t_size);
      *(reinterpret_cast<unsigned int*>(&block)) = idx;
      encrypt(reinterpret_cast<uint8_t*>(&block), key);
      return block;
    },
    transform_func
  );
}

// ===========================================================================================================================

template<typename scalar_t, typename uint_t>
void random_kernel_helper_fp(TensorIterator& iter, uint8_t* key) {
  block_cipher_helper<scalar_t, uint_t>(iter, key,
    [] __device__ (DummyRNG<1>* generator) -> scalar_t {
      if (std::is_same<scalar_t, double>::value) {
        return static_cast<scalar_t>(generator->random64() % static_cast<uint64_t>((1ULL << std::numeric_limits<scalar_t>::digits) + 1));
      } else {
        return static_cast<scalar_t>(generator->random() % static_cast<uint64_t>((1ULL << std::numeric_limits<scalar_t>::digits) + 1));
      }
    }
  );
}

template<typename scalar_t, typename uint_t>
void random_kernel_helper_int(TensorIterator& iter, uint8_t* key) {
  block_cipher_helper<scalar_t, uint_t>(iter, key,
    [] __device__ (DummyRNG<1>* generator) -> scalar_t {
      if (std::is_same<scalar_t, long>::value) {
        return static_cast<scalar_t>(generator->random64() % (static_cast<uint64_t>(std::numeric_limits<scalar_t>::max()) + 1));
      } else {
        return static_cast<scalar_t>(generator->random() % (static_cast<uint64_t>(std::numeric_limits<scalar_t>::max()) + 1));
      }
    }
  );
}

void random_kernel_helper_bool(TensorIterator& iter, uint8_t* key) {
  block_cipher_helper<bool, uint32_t>(iter, key,
    [] __device__ (DummyRNG<1>* generator) -> bool {
      return static_cast<bool>(generator->random() & 1);
    }
  );
}

void random_kernel(TensorIterator& iter, Tensor key_tensor) {
  const auto key = key_tensor.data_ptr<uint8_t>();
  if (isFloatingType(iter.dtype())) {
    AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "random_kernel_fp_cuda", [&] {
      if (std::is_same<scalar_t, double>::value) {
        random_kernel_helper_fp<scalar_t, uint64_t>(iter, key);
      } else {
        random_kernel_helper_fp<scalar_t, uint32_t>(iter, key);
      }
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/true)) {
    AT_DISPATCH_INTEGRAL_TYPES_AND(at::ScalarType::Bool, iter.dtype(), "random_kernel_int_cuda", [&] {
      if (std::is_same<scalar_t, int64_t>::value) {
        random_kernel_helper_int<scalar_t, uint64_t>(iter, key);
      } else if (std::is_same<scalar_t, bool>::value) {
        random_kernel_helper_bool(iter, key);
      } else {
        random_kernel_helper_int<scalar_t, uint32_t>(iter, key);
      }
    });
  }
}

template<typename RNG>
struct RandomKernel {
  void operator()(TensorIterator& iter, Generator generator) {
    random_kernel(iter, key_tensor(generator));
  }
};

Tensor& random_(Tensor& self, Generator generator) {
  return native::templates::random_impl<RandomKernel, CUDA_CSPRNG_GeneratorImpl>(self, generator);
}

// ===========================================================================================================================

// Using DistAccumType in accumulate types for distributions.
// Note: Ideally we'd be using ATen/AccumulateType.h but looks
// like the there is some inconsistency in how accumulate types
// are mapped currently, e.g. for the cpu side, float is mapped
// to double.
template <typename T>
struct DistAccumType {  };

#if defined(__HIPCC__) || defined(__HIPCC__)
template <> struct DistAccumType<half> { using type = float; };
#endif
template <> struct DistAccumType<Half> { using type = float; };
template <> struct DistAccumType<float> { using type = float; };
template <> struct DistAccumType<double> { using type = double; };

template <typename T>
using dist_acctype = typename DistAccumType<T>::type;

// Constants for uniform distribution
// doubles have 52 bits of mantissa (fractional part)
constexpr uint64_t DOUBLE_MASK = (1ULL << std::numeric_limits<double>::digits) - 1;
constexpr double DOUBLE_DIVISOR = 1.0 / (1ULL << std::numeric_limits<double>::digits);

// floats have 23 bits of mantissa (fractional part)
constexpr uint32_t FLOAT_MASK = (1 << std::numeric_limits<float>::digits) - 1;
constexpr float FLOAT_DIVISOR = 1.0f / (1 << std::numeric_limits<float>::digits);

template <typename T>
struct uniform_real_distribution {

  inline __device__ uniform_real_distribution(T a_in, T b_in) {
    // TORCH_CHECK(a_in <= b_in);
    // TORCH_CHECK(b_in-a_in <= std::numeric_limits<T>::max());
    a = a_in;
    b = b_in;
  }

  template <typename RNG>
  inline __device__ dist_acctype<T> operator()(RNG* generator){
    dist_acctype<T> x;
    if(std::is_same<T, double>::value) {
      x = (generator->random64() & DOUBLE_MASK) * DOUBLE_DIVISOR;
    } else {
      x = (generator->random() & FLOAT_MASK) * FLOAT_DIVISOR;
    }
    return (x * (b - a) + a);
  }

  private:
    T a;
    T b;
};

template<typename scalar_t, typename uint_t>
void uniform_kernel_helper_fp(TensorIterator& iter, uint8_t* key, scalar_t from, scalar_t to) {
  block_cipher_helper<scalar_t, uint_t>(iter, key,
    [from, to] __device__ (DummyRNG<1>* generator) -> scalar_t {
      uniform_real_distribution<scalar_t> uniform(from, to);
      return uniform(generator);
    }
  );
}

void uniform_kernel(TensorIterator& iter, Tensor key_tensor, double from, double to) {
  const auto key = key_tensor.data_ptr<uint8_t>();
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "uniform_kernel_cuda", [&] {
    if (std::is_same<scalar_t, double>::value) {
      uniform_kernel_helper_fp<scalar_t, uint64_t>(iter, key, from, to);
    } else {
      uniform_kernel_helper_fp<scalar_t, uint32_t>(iter, key, from, to);
    }
  });
}

template<typename RNG>
struct UniformKernel {
  void operator()(TensorIterator& iter, double from, double to, Generator generator) {
    uniform_kernel(iter, key_tensor(generator), from, to);
  }
};

Tensor& uniform_(Tensor& self, double from, double to, Generator generator) {
  return at::native::templates::uniform_impl_<UniformKernel, CUDA_CSPRNG_GeneratorImpl>(self, from, to, generator);
}

// ===========================================================================================================================

/**
 * Samples a normal distribution using the Box-Muller method
 * Takes mean and standard deviation as inputs
 * Note that Box-muller method returns two samples at a time.
 * Hence, we cache the "next" sample in the CPUGeneratorImpl class.
 */
 template <typename T>
 struct normal_distribution {
 
  inline __device__ normal_distribution(T mean_in, T stdv_in) {
  //  TORCH_CHECK(stdv_in > 0);
    mean = mean_in;
    stdv = stdv_in;
  }
 
  template <typename RNG>
  inline dist_acctype<T> __device__ operator()(RNG* generator) {
    dist_acctype<T> ret;
    // return cached values if available
    // if (std::is_same<T, double>::value) {
    //   if (generator->next_double_normal_sample()) {
    //     ret = *(generator->next_double_normal_sample()) * stdv + mean;
    //     // reset c10::optional to null
    //     generator->set_next_double_normal_sample(c10::optional<double>());
    //     return ret;
    //   }
    // } else {
    //   if (generator->next_float_normal_sample()) {
    //     ret = *(generator->next_float_normal_sample()) * stdv + mean;
    //     // reset c10::optional to null
    //     generator->set_next_float_normal_sample(c10::optional<float>());
    //     return ret;
    //   }
    // }
    // otherwise generate new normal values
    uniform_real_distribution<T> uniform(0.0, 1.0);
    const dist_acctype<T> u1 = uniform(generator);
    const dist_acctype<T> u2 = uniform(generator);
    const dist_acctype<T> r = ::sqrt(static_cast<T>(-2.0) * ::log(static_cast<T>(1.0)-u2));
    const dist_acctype<T> theta = static_cast<T>(2.0) * static_cast<T>(M_PI) * u1;
    // if (std::is_same<T, double>::value) {
    //   dist_acctype<double> cache = r * ::sin(theta);
    //   generator->set_next_double_normal_sample(c10::optional<double>(cache));
    // } else {
    //   dist_acctype<float> cache = r * ::sin(theta);
    //   generator->set_next_float_normal_sample(c10::optional<float>(cache));
    // }
    ret = r * ::cos(theta) * stdv + mean;
    return ret;
  }
 
  private:
    T mean;
    T stdv;
};

template<typename scalar_t, typename uint_t>
void normal_kernel_helper_fp(TensorIterator& iter, scalar_t mean, scalar_t std, uint8_t* key) {
  block_cipher_helper<scalar_t, uint_t, 2>(iter, key,
    [mean, std] __device__ (DummyRNG<2>* generator) -> scalar_t {
      normal_distribution<scalar_t> normal(mean, std);
      return normal(generator);
    }
  );
}

void normal_kernel(Tensor& self, double mean, double std, Tensor key_tensor) {
  const auto key = key_tensor.data_ptr<uint8_t>();
  auto iter = at::TensorIterator::nullary_op(self);
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "normal_kernel_cuda", [&] {
    if (std::is_same<scalar_t, double>::value) {
      normal_kernel_helper_fp<scalar_t, uint64_t>(iter, mean, std, key);
    } else {
      normal_kernel_helper_fp<scalar_t, uint32_t>(iter, mean, std, key);
    }
  });
}

template<typename RNG>
struct NormalKernel {
  void operator()(Tensor& self, double mean, double std, Generator generator) {
    normal_kernel(self, mean, std, key_tensor(generator));
  }
};

Tensor& normal_(Tensor& self, double mean, double std, Generator gen) {
  return at::native::templates::normal_impl_<NormalKernel, CUDA_CSPRNG_GeneratorImpl>(self, mean, std, gen);
}

// ===========================================================================================================================

Generator create_CUDA_CSPRNG_Generator() {
  return make_generator<CUDA_CSPRNG_GeneratorImpl>();
}

void registerOps() {
  static auto registry = torch::import()
    .impl_UNBOXED("aten::random_", DispatchKey::CustomRNGKeyId, random_)
    .impl_UNBOXED("aten::uniform_", DispatchKey::CustomRNGKeyId, uniform_)
    .impl_UNBOXED("aten::normal_", DispatchKey::CustomRNGKeyId, normal_);
}
  
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("registerOps", &registerOps);
  m.def("create_CUDA_CSPRNG_Generator", &create_CUDA_CSPRNG_Generator);
}
